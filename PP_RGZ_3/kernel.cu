#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

hipError_t addWithCuda(long long value);

using namespace std;

__device__ bool Prime(long long n)
{
	for (int i = 2; i <= sqrt((double)n); i++)
		if (n%i == 0)
			return false;
	return true;
}

__global__ void addKernel(long long from, long long *a, int *output, int cudaCores)
{
	const long long current = threadIdx.x + from + cudaCores * blockIdx.x;

	long long outPos = current - from;

	output[outPos] = 0;

	if (a[0]%current == 0 && Prime(current))
	{
		while (a[0]%current == 0)
		{
			output[outPos] = 1;
		}
	}

	/*for (int k = 0; k < size; k++)
	{
		if (Prime(a[0], from, to) == true) {}
		else continue;
	}*/
}

int main()
{
	/*const int arraySize = 500000;
	int *a = new int[arraySize];
	int *c = new int[arraySize / 2];

	for (int c = 1; c < arraySize; c++)
	{
		a[c - 1] = c;
	}*/

	long long value;

	cout << "Write value: "; cin >> value;

	// Add vectors in parallel.
	auto begin = chrono::high_resolution_clock::now();
	hipError_t cudaStatus = addWithCuda(value);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	auto end = chrono::high_resolution_clock::now();

	/*int i = 0;
	while (c[i] > 0)
	{
		cout << c[i] << endl;
		i++;
	}*/

	cout << "Work time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << endl;
	system("pause");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(long long value)
{
	long long *dev_a = 0;

	int cudaCores = 100;
	int blocksCount = 1;

	long long from = 2;
	//long long to = sqrt((double)value);
	const long long bufferSize = value - from;
	const long long blockCount = (bufferSize / cudaCores) + (bufferSize%cudaCores == 0 ? 0 : 1);

	if (bufferSize < cudaCores)
	{
		cudaCores = bufferSize;
	}

	/*long long *from = new long long[bufferSize];
	long long *to = new long long[bufferSize];

	long long step = (to - from) / cudaCores;

	for (int i = 0; i < bufferSize; i++) // ��������� �� � ��
	{
		from[&i] = from + step * i;
		to[&i] = from + step * i + step;
	}*/

	int *output = new int[bufferSize];
	int *dev_output;

	hipError_t cudaStatus;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	/*cudaStatus = hipMalloc((void**)&dev_c, 1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}*/

	cudaStatus = hipMalloc((void**)&dev_a, 1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_output, bufferSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, &value, 1, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//int threadsPerBlock = 55;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	// Launch a kernel on the GPU with one thread for each element.

	hipEventRecord(start, 0);

	addKernel <<< blocksCount, cudaCores >>> (from, dev_a, dev_output, cudaCores);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	hipEventRecord(stop, 0);
	float time = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, 1, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	/*cudaStatus = hipMemcpy(c, dev_c, 1, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

Error:
	hipFree(dev_output);
	hipFree(dev_a);

	cout << "Work time: " << time << endl;
	system("pause");

	return cudaStatus;
}