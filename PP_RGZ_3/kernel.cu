#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

hipError_t addWithCuda(long long value);

using namespace std;

__global__ void addKernel(long long from, long long a, char *output, int cudaCores)
{
	const long long current = threadIdx.x + from + cudaCores * blockIdx.x;

	long long outPos = current - from;

	output[outPos] = 0;

	if (a % current == 0) output[outPos] = -1;
	else output[outPos] = 1;

	/*for (int i = from; i < sqrt((double)a); i++)
	{
		if (a % i == 0) output[outPos] = -1;
		else output[outPos] = 1;
	}*/
}

int main()
{
	long long value;

	cout << "Write value: "; cin >> value;

	// Add vectors in parallel.
	auto begin = chrono::high_resolution_clock::now();
	hipError_t cudaStatus = addWithCuda(value);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	auto end = chrono::high_resolution_clock::now();

	cout << "Work time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << endl;

	system("pause");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(long long value)
{
	int cudaCores = 1000;

	long long from = 2;
	const long long bufferSize = value - from;
	const long long blockCount = (bufferSize / cudaCores) + (bufferSize%cudaCores == 0 ? 0 : 1);

	if (bufferSize < cudaCores)
	{
		cudaCores = bufferSize;
	}

	char *output = new char[bufferSize];
	char *dev_output;

	hipError_t cudaStatus;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**)&dev_output, bufferSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	// Launch a kernel on the GPU with one thread for each element.

	hipEventRecord(start, 0);

	addKernel <<< blockCount, cudaCores >>> (from, value, dev_output, cudaCores);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	hipEventRecord(stop, 0);
	float time = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_output, 1, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_output);

	cout << (int)output[0] << endl;
	if ((int)output[0] > 0)	cout << "Chislo " << value << " prostoe" << endl;
	else cout << "Chislo " << value << " ne prostoe" << endl;
	
	/*while (y < bufferSize)
	{
		cout << (int)output[y] << endl;
		y++;
	}*/

	cout << "Work time: " << time << endl;
	system("pause");

	return cudaStatus;
}