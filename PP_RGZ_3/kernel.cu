#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, unsigned int size);

using namespace std;

__device__ bool Prime(long long n)
{
	for (int i = 2; i <= sqrt((double)n); i++)
		if (n%i == 0)
			return false;
	return true;
}

__global__ void addKernel(int *c, const int *a, int size)
{
	int j = 0;
	//int size = 10000;

	for (int k = 0; k < size; k++)
	{
		if (Prime(a[k]) == true) { c[j] = a[k]; j++; }
		else continue;
	}
}

int main()
{
	const int arraySize = 50000;
	int a[arraySize];
	int c[arraySize];

	for (int c = 1; c < arraySize; c++)
	{
		a[c - 1] = c;
	}

	// Add vectors in parallel.

	hipError_t cudaStatus = addWithCuda(c, a, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	int i = 0;
	while (c[i] > 0)
	{
		cout << c[i] << endl;
		i++;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, unsigned int size)
{
	int *dev_a = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	// Launch a kernel on the GPU with one thread for each element.
	auto begin = chrono::high_resolution_clock::now();
	addKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, size);
	auto end = chrono::high_resolution_clock::now();

	cout << "Work time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << endl;
	system("pause");

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	return cudaStatus;
}