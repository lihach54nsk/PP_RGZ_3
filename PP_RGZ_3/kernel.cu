#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, unsigned int size);

using namespace std;

__device__ bool Prime(long long n)
{
	for (int i = 2; i <= sqrt((double)n); i++)
		if (n%i == 0)
			return false;
	return true;
}

__global__ void addKernel(int *c, const int *a, int size)
{
	int j = 0;

	for (int k = 0; k < size; k++)
	{
		if (Prime(a[k]) == true) { c[j] = a[k]; j++; }
		else continue;
	}
}

int main()
{
	const int arraySize = 400000;
	int *a = new int[arraySize];
	int *c = new int[arraySize];

	for (int c = 1; c < arraySize; c++)
	{
		a[c - 1] = c;
	}

	// Add vectors in parallel.
	auto begin = chrono::high_resolution_clock::now();
	hipError_t cudaStatus = addWithCuda(c, a, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	auto end = chrono::high_resolution_clock::now();

	int i = 0;
	while (c[i] > 0)
	{
		cout << c[i] << endl;
		i++;
	}

	cout << "Work time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << endl;
	system("pause");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, unsigned int size)
{
	int *dev_a = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//int threadsPerBlock = 55;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	// Launch a kernel on the GPU with one thread for each element.

	hipEventRecord(start, 0);

	addKernel <<< 10, 100 >>> (dev_c, dev_a, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	hipEventRecord(stop, 0);
	float time = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);

	cout << "Work time: " << time << endl;
	system("pause");

	return cudaStatus;
}